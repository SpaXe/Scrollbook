#include "hip/hip_runtime.h"
///
/// Author: Xavier Ho (contact@xavierho.com)
///
#include <iostream>
#include <cmath>
#include <string>
#include <string.h>
#ifdef _WIN32
  #include "GL/glew.h"
  #include "GL/glfw.h"
#else
  #include <GL/glew.h>
  #include <GL/glfw.h>
#endif
#include "Timer.h"
#include "Mandelbrot.h"
using namespace std;

#ifdef DEBUG
  #define CUDA_ERROR(x) \
  { \
    hipError_t r = x; \
    if (r != hipSuccess) \
      { cout << #x << "\n" << r << ": " << hipGetErrorString(r) << endl; } \
  }
#else
  #define CUDA_ERROR(x) x
#endif

/// Basic implementation of Mandelbrot on CUDA
/// Completely unoptimised, and is only good for 1024x1024 resolution at this stage.
/// TODO: Look into CUDA textures
/// TODO: Test with OpenGL Shaders, and use that as a baseline benchmark speed for optimising this kernel.
__global__ void mandelbrot(unsigned char * data, int width, int height, float tx, float ty, float scale, int limit)
{
  unsigned int u = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int v = blockIdx.y * blockDim.y + threadIdx.y;

  /// Each thread renders 16 pixels
  for (int i = 0; i < 4; ++i) {
    for (int j = 0; j < 4; ++j) {
      // This line is a little tricky to work out.
      unsigned int addr = v*width*3*4 + u*3*4 + i*width*3 + j*3;
      float cr = float(u*4 + j + tx) / width * scale;
      float ci = float(v*4 + i + ty) / height * scale;

      int c = 0;
      float tmp, x = 0, y = 0;
      /// Surely we can fold this? Don't have time, but perhaps a PDE can be derived here and find the
      /// difference equation to save on loops
      while ((x*x + y*y < 4) && (c++ < limit)) {
          tmp = x*x - y*y + cr;
          y = 2 * x * y + ci;
          x = tmp;
      }
      /// TODO: Colour remapping so we don't have ugly, static blue colours
      if (c >= limit) {
        data[addr]   =  (unsigned char) 0;
        data[addr+1] =  (unsigned char) 0;
        data[addr+2] =  (unsigned char) 0;
      } else {
        unsigned char value = (unsigned char)((float)c / limit * 255);
        data[addr]   = value;
        data[addr+1] = value / 2;
        data[addr+2] = value / 4;
      }
    }
  }
}


Mandelbrot::Mandelbrot(int width, int height)
{
  this->width = width;
  this->height = height;
  this->limit = 64;
  this->scale = 3.0f;
  this->tx = -width * 5 / 7.0f;
  this->ty = -height / 2.0f;
  this->data = new unsigned char[width * height * 3];
  memset(this->data, 0, sizeof(data));
  this->size = width * height * 3 * sizeof(unsigned char);
  CUDA_ERROR(hipMalloc((void **)&this->gpu_data, this->size));
}


Mandelbrot::~Mandelbrot()
{
  delete[] this->data;
  hipFree(gpu_data);
}


void Mandelbrot::render()
{
  /// Create a new window and setup OpenGL context
  glfwInit();
  glfwOpenWindow(width, height, 8, 8, 8, 0, 8, 0, GLFW_WINDOW);

  /// Define our view and a 2D texture to be mapped onto a full-screen quad
  glMatrixMode(GL_PROJECTION);
  glLoadIdentity();
  glOrtho(0, width, 0, height, -1.0, 1.0);
  glViewport(0, 0, width, height);
  glClearColor(0, 0, 0, 0);
  glColor3f(1, 1, 1);

  glEnable(GL_TEXTURE_2D);
  glGenTextures(1, &texture_id);
  glBindTexture(GL_TEXTURE_2D, texture_id);
  glTexParameterf(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
  glTexParameterf(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
  glTexParameterf(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
  glTexParameterf(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);
  glTexImage2D(GL_TEXTURE_2D,
                0, 3, width, height, 0, GL_BGR_EXT, 
                GL_UNSIGNED_BYTE, data);

  /// Reset the GPU buffer so we're sure the frame is up to date.
  CUDA_ERROR(hipMemcpy(gpu_data, data, this->size, hipMemcpyHostToDevice));
  
  while (running)
  {
    /// Timing is taken just with the kernel, because GLFW (similar to GLUT) is capped at 60 FPS.
    /// I should have used SDL instead, but good to learn another framework anyway
    timer.start();

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks(width/4/threadsPerBlock.x, height/4/threadsPerBlock.y);
    mandelbrot<<<numBlocks, threadsPerBlock>>>(gpu_data, width, height, tx, ty, scale, limit);
    CUDA_ERROR(hipDeviceSynchronize());
    CUDA_ERROR(hipMemcpy(data, gpu_data, this->size, hipMemcpyDeviceToHost));

    elapsed_time = timer.getMilliseconds();
    cout << elapsed_time << " ms" << endl;

    /// Draw a full-screen quad
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glClear(GL_COLOR_BUFFER_BIT);

    glTexSubImage2D(GL_TEXTURE_2D, 
                    0, 0, 0, width, height, GL_BGR_EXT, 
                    GL_UNSIGNED_BYTE, data);

    glBegin(GL_QUADS);
      glTexCoord2f(0, 0);
      glVertex3i(0, 0, 0);
        
      glTexCoord2f(1, 0);
      glVertex3i(width, 0, 0);
        
      glTexCoord2f(1, 1);
      glVertex3i(width, height, 0);
        
      glTexCoord2f(0, 1);
      glVertex3i(0, height, 0);
    glEnd();

    glfwSwapBuffers();
    glfwPollEvents();
    handle_inputs();
  }
}

void Mandelbrot::handle_inputs()
{
  if (glfwGetWindowParam(GLFW_OPENED) == GL_FALSE 
          || glfwGetKey(GLFW_KEY_ESC) == GLFW_PRESS)
    running = false;
  if (glfwGetKey('H') == GLFW_PRESS) {
    scale = 2.f;
    tx = -width * 2 / 3.0f;
    ty = -height / 3.0f;
  }
  if (glfwGetKey('W') == GLFW_PRESS)
    ty -= height >> 4;
  if (glfwGetKey('A') == GLFW_PRESS)
    tx += width >> 4;
  if (glfwGetKey('S') == GLFW_PRESS)
    ty += height >> 4;
  if (glfwGetKey('D') == GLFW_PRESS)
    tx -= width >> 4;
  if (glfwGetKey('Q') == GLFW_PRESS)
    scale += 0.25f;
  if (glfwGetKey('E') == GLFW_PRESS)
    scale -= 0.25f;
  if (glfwGetKey(']') == GLFW_PRESS) {
    limit /= 2;
    if (limit < 1) limit = 2;
  }
  if (glfwGetKey('[') == GLFW_PRESS) {
    limit *= 2;
    if (limit > 1024) limit = 1024;
  }
}

int main(int argc, char* argv[])
{    
  Mandelbrot m(1024, 1024);
  m.render();
  return EXIT_SUCCESS;
}
