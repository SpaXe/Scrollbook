#include "hip/hip_runtime.h"
///
/// Author: Xavier Ho (contact@xavierho.com)
///
#include <iostream>
#include <cmath>
#include <string>
#include <string.h>
#include "GL\glew.h"
#include "GL\glfw.h"
#include "Timer.h"
#include "Mandlebrot.h"
using namespace std;

#define CUDA_ERROR(x) \
{ \
  hipError_t r = x; \
  if (r != hipSuccess) \
    { cout << #x << "\n" << r << ": " << hipGetErrorString(r) << endl; } \
}

/// Basic implementation of Mandlebrot on CUDA
__global__ void mandlebrot(unsigned char * data, int width, int height, float tx, float ty, float scale, int limit)
{
  // Convert to normalised texture coordinates
  unsigned int u = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int v = blockIdx.y * blockDim.y + threadIdx.y;

  for (int i = 0; i < 4; ++i) {
    for (int j = 0; j < 4; ++j) {
      unsigned int addr = v*width*3*4 + u*3*4 + i*width*3 + j*3;
      float cr = float(u*4 + j + tx) / width * scale;
      float ci = float(v*4 + i + ty) / height * scale;

      int c = 0;
      float tmp, x = 0, y = 0;
      while ((x*x + y*y < 4) && (c++ < limit)) {
          tmp = x*x - y*y + cr;
          y = 2 * x * y + ci;
          x = tmp;
      }
      if (c >= limit) {
        data[addr]   =  (unsigned char) 0;
        data[addr+1] =  (unsigned char) 0;
        data[addr+2] =  (unsigned char) 0;
      } else {
        unsigned char value = (unsigned char)((float)c / limit * 255);
        data[addr]   = value;
        data[addr+1] = value / 2;
        data[addr+2] = value / 4;
      }
    }
  }
}


Mandlebrot::Mandlebrot(int width, int height)
{
  this->width = width;
  this->height = height;
  this->limit = 64;
  this->scale = 3.0f;
  this->tx = -width * 5 / 7.0f;
  this->ty = -height / 2.0f;
  this->data = new unsigned char[width * height * 3];
  memset(this->data, 0, sizeof(data));
  this->size = width * height * 3 * sizeof(unsigned char);
  CUDA_ERROR(hipMalloc((void **)&this->gpu_data, this->size));
}

Mandlebrot::~Mandlebrot()
{
  delete[] this->data;
  hipFree(gpu_data);
}

void Mandlebrot::render()
{
  glfwInit();
  glfwOpenWindow(width, height, 8, 8, 8, 0, 8, 0, GLFW_WINDOW);

  glMatrixMode(GL_PROJECTION);
  glLoadIdentity();
  glOrtho(0, width, 0, height, -1.0, 1.0);
  glViewport(0, 0, width, height);
  glClearColor(0, 0, 0, 0);
  glColor3f(1, 1, 1);

  glEnable(GL_TEXTURE_2D);
  glGenTextures(1, &texture_id);
  glBindTexture(GL_TEXTURE_2D, texture_id);
  glTexParameterf(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
  glTexParameterf(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
  glTexParameterf(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
  glTexParameterf(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);
  glTexImage2D(GL_TEXTURE_2D,
                0, 3, width, height, 0, GL_BGR_EXT, 
                GL_UNSIGNED_BYTE, data);

  CUDA_ERROR(hipMemcpy(gpu_data, data, this->size, hipMemcpyHostToDevice));

  timer.start();
  while (running)
  {
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks(width/4/threadsPerBlock.x, height/4/threadsPerBlock.y);
    mandlebrot<<<numBlocks, threadsPerBlock>>>(gpu_data, width, height, tx, ty, scale, limit);
    CUDA_ERROR(hipDeviceSynchronize());
    CUDA_ERROR(hipMemcpy(data, gpu_data, this->size, hipMemcpyDeviceToHost));

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glClear(GL_COLOR_BUFFER_BIT);

    glTexSubImage2D(GL_TEXTURE_2D, 
                    0, 0, 0, width, height, GL_BGR_EXT, 
                    GL_UNSIGNED_BYTE, data);

    glBegin(GL_QUADS);
      glTexCoord2f(0, 0);
      glVertex3i(0, 0, 0);
        
      glTexCoord2f(1, 0);
      glVertex3i(width, 0, 0);
        
      glTexCoord2f(1, 1);
      glVertex3i(width, height, 0);
        
      glTexCoord2f(0, 1);
      glVertex3i(0, height, 0);
    glEnd();

    glfwSwapBuffers();
    glfwPollEvents();
    handle_inputs();
    elapsed_time = timer.getMilliseconds();
    cout << elapsed_time << " ms" << endl;
    timer.start();
  }
}

void Mandlebrot::handle_inputs()
{
  if (glfwGetWindowParam(GLFW_OPENED) == GL_FALSE 
          || glfwGetKey(GLFW_KEY_ESC) == GLFW_PRESS)
    running = false;
  if (glfwGetKey('H') == GLFW_PRESS) {
    scale = 2.f;
    tx = -width * 2 / 3.0f;
    ty = -height / 3.0f;
  }
  if (glfwGetKey('W') == GLFW_PRESS)
    ty -= height >> 4;
  if (glfwGetKey('A') == GLFW_PRESS)
    tx += width >> 4;
  if (glfwGetKey('S') == GLFW_PRESS)
    ty += height >> 4;
  if (glfwGetKey('D') == GLFW_PRESS)
    tx -= width >> 4;
  if (glfwGetKey('Q') == GLFW_PRESS)
    scale += 0.25f;
  if (glfwGetKey('E') == GLFW_PRESS)
    scale -= 0.25f;
  if (glfwGetKey(']') == GLFW_PRESS) {
    limit /= 2;
    if (limit < 1) limit = 2;
  }
  if (glfwGetKey('[') == GLFW_PRESS) {
    limit *= 2;
    if (limit > 1024) limit = 1024;
  }
}

int main(int argc, char* argv[])
{    
  Mandlebrot m(1024, 1024);
  m.render();
  return EXIT_SUCCESS;
}


/*
void TextureRenderer::handleMouse()
{
  glfwGetMousePos(&mx, &my);
  drag = glfwGetMouseButton(GLFW_MOUSE_BUTTON_LEFT) == GLFW_PRESS;
  zoom = glfwGetMouseButton(GLFW_MOUSE_BUTTON_RIGHT) == GLFW_PRESS;
}


void TextureRenderer::handleMouseMove()
{
  int x, y;
  glfwGetMousePos(&x, &y);
  if (drag) {
    tx += (mx - x);
    ty += (y - my);
    mx = x;
    my = y;
  } else if (zoom) {
    if (mx - x < 0)
      scale -= 0.1f;
    else if (mx - x > 0)
      scale += 0.1f;
    else
    {} // Nothing
  }
}*/



/*
TextureRenderer::TextureRenderer(int width, int height)
  : bbox(0, 0, width, height)
{
  this->width = width;
  this->height = height;
  data = new unsigned char[width * height * 3];
  
  scale = 2.f;
  tx = -width*2/3;
  ty = -height/3;
  limit = 32;

  mx = my = 0;
  drag = zoom = false;
  running = true;
  rendererType = 1;
  
  elapsed_timer = 0.0;
  modeText = "1: CPU (single thread)";
  computeText = "Time/fractal: ";
  fpsText = "Time/frame: ";

  glfwInit();
}*/

/*
void TextureRenderer::drawText(int x, int y, const string& text)
{
  // glRasterPos2i(x, y);
  // for (int i = 0; i < (int)text.size(); i++)
    //glutBitmapCharacter(GLUT_BITMAP_8_BY_13, text[i]);
  // TODO: If time allows, print to screen.
  cout << text << endl;
}*/