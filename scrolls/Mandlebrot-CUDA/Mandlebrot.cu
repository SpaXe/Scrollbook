///
/// Author: Xavier Ho (contact@xavierho.com)
///
#include <iostream>
#include <cmath>
#include <string>
#include "GL\glew.h"
#include "GL\glfw.h"
#include "Mandlebrot.h"
using namespace std;

Mandlebrot::Mandlebrot(int width, int height)
{
  this->width = width;
  this->height = height;
  this->limit = 64;
  this->scale = 3.0f;
  this->tx = -width * 5 / 7.0f;
  this->ty = -height / 2.0f;
  this->data = new unsigned char[width * height * 3];
}

Mandlebrot::~Mandlebrot()
{
  delete[] this->data;
}

float Mandlebrot::pixel_at(float cr, float ci)
{
  float x = 0, y = 0;
  float tmp;
  int i = 0;
  while ((x*x + y*y < 4) && (i++ < limit)) {
      tmp = x*x - y*y + cr;
      y = 2 * x * y + ci;
      x = tmp;
  }
  if (i >= limit)
    return 0.0f;
  return i / (float)(limit);
}

void Mandlebrot::render()
{
  for (int j = 0; j < height; ++j) {
    for (int i = 0; i < width; ++i) {  
      float x = (float)(i + tx) / width * scale;
      float y = (float)(j + ty) / height * scale;
      unsigned char value = (unsigned char)(pixel_at(x, y) * 255);
      data[j*height*3+i*3] = value;
      data[j*height*3+1+i*3] = value >> 1;
      data[j*height*3+2+i*3] = value >> 2;
    }
  }
}

void Mandlebrot::handle_inputs()
{
  if (glfwGetKey('H') == GLFW_PRESS) {
    scale = 2.f;
    tx = -width * 2 / 3.0f;
    ty = -height / 3.0f;
  }
  if (glfwGetKey('W') == GLFW_PRESS)
    ty -= height >> 4;
  if (glfwGetKey('A') == GLFW_PRESS)
    tx += width >> 4;
  if (glfwGetKey('S') == GLFW_PRESS)
    ty += height >> 4;
  if (glfwGetKey('D') == GLFW_PRESS)
    tx -= width >> 4;
  if (glfwGetKey('Q') == GLFW_PRESS)
    scale += 0.25f;
  if (glfwGetKey('E') == GLFW_PRESS)
    scale -= 0.25f;
  if (glfwGetKey('-') == GLFW_PRESS) {
    limit /= 2;
    if (limit < 1) limit = 2;
  }
  if (glfwGetKey('+') == GLFW_PRESS) {
    limit *= 2;
    if (limit > 1024) limit = 1024;
  }
}

int main(int argc, char* argv[])
{    
  Mandlebrot m(1024, 1024);
  return EXIT_SUCCESS;
}


/*
void TextureRenderer::handleMouse()
{
  glfwGetMousePos(&mx, &my);
  drag = glfwGetMouseButton(GLFW_MOUSE_BUTTON_LEFT) == GLFW_PRESS;
  zoom = glfwGetMouseButton(GLFW_MOUSE_BUTTON_RIGHT) == GLFW_PRESS;
}


void TextureRenderer::handleMouseMove()
{
  int x, y;
  glfwGetMousePos(&x, &y);
  if (drag) {
    tx += (mx - x);
    ty += (y - my);
    mx = x;
    my = y;
  } else if (zoom) {
    if (mx - x < 0)
      scale -= 0.1f;
    else if (mx - x > 0)
      scale += 0.1f;
    else
    {} // Nothing
  }
}*/



/*
TextureRenderer::TextureRenderer(int width, int height)
  : bbox(0, 0, width, height)
{
  this->width = width;
  this->height = height;
  data = new unsigned char[width * height * 3];
  
  scale = 2.f;
  tx = -width*2/3;
  ty = -height/3;
  limit = 32;

  mx = my = 0;
  drag = zoom = false;
  running = true;
  rendererType = 1;
  
  elapsed_timer = 0.0;
  modeText = "1: CPU (single thread)";
  computeText = "Time/fractal: ";
  fpsText = "Time/frame: ";

  glfwInit();
}*/

/*
void TextureRenderer::drawText(int x, int y, const string& text)
{
  // glRasterPos2i(x, y);
  // for (int i = 0; i < (int)text.size(); i++)
    //glutBitmapCharacter(GLUT_BITMAP_8_BY_13, text[i]);
  // TODO: If time allows, print to screen.
  cout << text << endl;
}*/